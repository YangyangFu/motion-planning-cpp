
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

//kernel function to add two arrays in GPU -> __global__ keyword
__global__
void add(int n, float* x, float* y) {
    int index = threadIdx.x; // thread index within the block
    int stride = blockDim.x; // total number of threads in a block

    // for each thread, run the following loop
    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

int main() {
    int N = 1 << 20; // ~1M elements 2^(20)

    // allocate unified memory -> accessible from both CPU and GPU
    float *x, *y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // add the arrays on the GPUs
    // <<<number of blocks per grid, number of threads per block>>>
    add<<<1, 256>>>(N, x, y);
    
    // wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // verify the result -> all values in y should be 3.0f
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }

    std::cout << "Max error: " << maxError << std::endl;

    // free unified memory
    hipFree(x);
    hipFree(y);

    return 0;
}